/*
 * Copyright (c) 2024 NVIDIA Corporation
 *
 * Licensed under the Apache License Version 2.0 with LLVM Exceptions
 * (the "License"); you may not use this file except in compliance with
 * the License. You may obtain a copy of the License at
 *
 *   https://llvm.org/LICENSE.txt
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "ustdex/ustdex.hpp"

#include <cstdio>
#include <string>

using namespace ustdex;

struct sink {
  using receiver_concept = receiver_t;

  USTDEX_HOST_DEVICE void set_value() noexcept {
  }

  USTDEX_HOST_DEVICE void set_value(int a) noexcept {
    std::printf("%d\n", a);
  }

  template <class... As>
  USTDEX_HOST_DEVICE void set_value(As &&...) noexcept {
    std::printf("%s\n", "In sink::set_value(auto&&...)");
  }

  USTDEX_HOST_DEVICE void set_error(std::exception_ptr eptr) noexcept {
    std::printf("Error\n");
  }

  USTDEX_HOST_DEVICE void set_stopped() noexcept {
  }
};

struct _inline_scheduler {
  using scheduler_concept = scheduler_t;

  template <class Rcvr>
  struct _opstate_t {
    using operation_state_concept = operation_state_t;
    Rcvr rcvr;

    USTDEX_HOST_DEVICE void start() noexcept {
      ustdex::set_value(static_cast<Rcvr&&>(rcvr));
    }
  };

  struct _sndr_t {
    using sender_concept = sender_t;

    auto get_completion_signatures(_ignore_t = {}) const
      -> completion_signatures<set_value_t()>;

    template <class Rcvr>
    USTDEX_HOST_DEVICE auto connect(Rcvr rcvr) const noexcept {
      return _opstate_t<Rcvr>{rcvr};
    }
  };

  USTDEX_HOST_DEVICE _sndr_t schedule() noexcept {
    return {};
  }
};

template <class>
[[deprecated]]
void print() {
}

USTDEX_HOST_DEVICE void _main() {
  auto s = start_on(
    _inline_scheduler(),
    then(
      just(1, 2, 3),
      []USTDEX_HOST_DEVICE(int a, int b, int c) {
        std::printf("%d %d %d\n", a, b, c);
        return a + b + c;
      }
    )
  );

  auto o = connect(s, sink{});
  start(o);

  constexpr auto just_plus_one = [](int a) { return just(a + 1); };
  auto s3 = let_value(just(42), just_plus_one);
  auto o3 = connect(s3, sink{});
  start(o3);

  auto s4 = just(42) | then([](int){}) | upon_error([](auto){ /*return 42;*/ });
  auto s5 = when_all(std::move(s4), just(42, 42), just(+""));
  auto o5 = connect(std::move(s5), sink{});
  o5.start();
  // using X = completion_signatures_of_t<decltype(s5)>;
  // print<X>();
}

int main() {
  _main();
}